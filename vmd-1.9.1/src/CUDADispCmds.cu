/***************************************************************************
 *cr                                                                       
 *cr            (C) Copyright 1995-2011 The Board of Trustees of the           
 *cr                        University of Illinois                       
 *cr                         All Rights Reserved                        
 *cr                                                                   
 ***************************************************************************/

/***************************************************************************
 * RCS INFORMATION:
 *
 *	$RCSfile: CUDADispCmds.cu,v $
 *	$Author: johns $	$Locker:  $		$State: Exp $
 *	$Revision: 1.4 $	$Date: 2011/12/23 23:56:19 $
 *
 ***************************************************************************
 * DESCRIPTION:
 *
 * DispCmds - different display commands which take data and put it in
 *	a storage space provided by a given VMDDisplayList object.
 *
 * Notes:
 *	1. All coordinates are stored as 3 points (x,y,z), even if meant
 * for a 2D object.  The 3rd coord for 2D objects will be ignored.
 ***************************************************************************/

#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "Scene.h"
#include "DispCmds.h"
#include "utilities.h"
#include "Matrix4.h"
#include "VMDDisplayList.h"


//*************************************************************
// draw a mesh consisting of vertices, facets, colors, normals etc.
void DispCmdTriMesh::cuda_putdata(const float * vertices_d,
                                  const float * normals_d,
                                  const float * colors_d,
                                  int num_facets,
                                  VMDDisplayList * dobj) {
  // make a triangle mesh (no strips)
  DispCmdTriMesh *ptr;
  if (colors_d == NULL) {
    ptr = (DispCmdTriMesh *)
                (dobj->append(DTRIMESH_C3F_N3F_V3F, sizeof(DispCmdTriMesh) +
                                        sizeof(float) * num_facets * 3 * 6));
  } else {
    ptr = (DispCmdTriMesh *)
                (dobj->append(DTRIMESH_C3F_N3F_V3F, sizeof(DispCmdTriMesh) +
                                        sizeof(float) * num_facets * 3 * 9));
  }

  if (ptr == NULL)
    return;

  ptr->numverts=num_facets * 3;
  ptr->numfacets=num_facets;

  float *c=NULL, *n=NULL, *v=NULL;
  if (colors_d == NULL) {
    ptr->pervertexcolors=0;
    ptr->getpointers(n, v);
  } else {
    ptr->pervertexcolors=1;
    ptr->getpointers(c, n, v);
    hipMemcpy(c, colors_d,   ptr->numverts * 3 * sizeof(float), hipMemcpyDeviceToHost);
  }

  hipMemcpy(n, normals_d,  ptr->numverts * 3 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(v, vertices_d, ptr->numverts * 3 * sizeof(float), hipMemcpyDeviceToHost);
}



